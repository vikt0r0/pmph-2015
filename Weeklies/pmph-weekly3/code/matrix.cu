#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <stdlib.h>
#include <math.h>
#include <sys/time.h>
#include "matrix.cu.h"
#ifdef __HIPCC__
#include "matrix_kernels.cu.h"
#include <hip/hip_runtime.h>
#endif

#define EPSILON 0.001
#define MATRIX_SIZE 16
#define TILE_SIZE 32
#define NUM_THREADS_BLOCK 8

#define RAND_FLOAT(min,max) (min + static_cast <float> (rand()) /(static_cast <float> (RAND_MAX/(max-min))))
#define APPROX_EQUAL(a,b,epsilon) (fabs(a - b) <= ( (fabs(a) < fabs(b) ? fabs(b) : fabs(a)) * epsilon))

struct timeval t_start, t_end, t_diff;

template <typename T>
void matrix_set_element(matrix_t<T> mat, int i, int j, T value) {
  mat.elements[i*mat.width+j] = value;
}

template <typename T>
T matrix_get_element(matrix_t<T> mat, int i, int j) {
  return mat.elements[i*mat.width+j];
}

template <typename T>
bool matrix_is_equal(matrix_t<T> a, matrix_t<T> b) {
  if (a.width != b.width || a.height != b.height)
    return false;

  bool equal = true;

  for (int i = 0; i < a.height; ++i) {
    for (int j = 0; j < a.width; ++j) {
      if (matrix_get_element(a,i,j) != matrix_get_element(b,i,j)) {
        equal = false;
        break;
      }
    }
  }
  return equal;
}

void print_matrix(matrix_t<float> m) {
  int width = m.width;
  int height = m.height;
  for (int i=0; i < height; ++i) {
    for (int j=0; j < width; ++j) {
      printf("%06.3f\t", m.elements[i*width+j]);
    }
    printf("\n");
  }
}

template <>
bool matrix_is_equal(matrix_t<float> a, matrix_t<float> b) {
  if (a.width != b.width || a.height != b.height)
    return false;

  bool equal = true;

  for (int i = 0; i < a.height; ++i) {
    for (int j = 0; j < a.width; ++j) {
      if (!APPROX_EQUAL(matrix_get_element(a,i,j), matrix_get_element(b,i,j), EPSILON)) {
        equal = false;
        break;
      }
    }
  }
  return equal;
}

void matrix_fill_random_float(matrix_t<float> mat, float min, float max) {
  for (int i = 0; i < mat.height; ++i)
    for (int j = 0; j < mat.width; ++j)
      matrix_set_element(mat, i, j, RAND_FLOAT(min,max));
}

template <typename T>
void matrix_transpose_seq(matrix_t<T> out, matrix_t<T> in) {
  for (int i = 0; i < in.height; ++i) {
    for (int j = 0; j < in.width; ++j) {
      matrix_set_element(out, j, i, matrix_get_element(in, i, j));
    }
  }
}

template <typename T>
void matrix_transpose_omp(matrix_t<T> out, matrix_t<T> in) {
  #if defined(_OPENMP)
  #pragma omp parallel for
  #endif
  for (int i = 0; i < in.height; ++i) {
    for (int j = 0; j < in.width; ++j) {
      matrix_set_element(out, j, i, matrix_get_element(in, i, j));
    }
  }
}

template <typename T>
void matrix_transpose_cuda_naive(const unsigned int block_size, matrix_t<T> out, const matrix_t<T> in) {
  #ifndef __HIPCC__
    // If the CUDA compiler is not used, fall back to OMP implementation.
    matrix_transpose_omp<T>(out, in);
  #else
    // Set up and invoke kernel
    unsigned int num_blocks_x, num_blocks_y;

    num_blocks_x = (in.width  + block_size - 1) / block_size;
    num_blocks_y = (in.height + block_size - 1) / block_size;

    dim3 blockDim(block_size, block_size);
    dim3 gridDim(num_blocks_x, num_blocks_y);

    matrix_transpose_naive_kernel<T><<< gridDim, blockDim >>>(out, in);
    hipDeviceSynchronize();
  #endif
}

template <typename T>
void matrix_transpose_cuda_tiled(matrix_t<T> out, const matrix_t<T> in) {
  #ifndef __HIPCC__
    // If the CUDA compiler is not used, fall back to OMP implementation.
    matrix_transpose_omp<T>(out, in);
  #else
    // Set up and invoke kernel
    unsigned int num_blocks_x, num_blocks_y;

    num_blocks_x = (in.width + TILE_SIZE  - 1) / TILE_SIZE;
    num_blocks_y = (in.height + TILE_SIZE  - 1) / TILE_SIZE;
 
    dim3 blockDim(TILE_SIZE, TILE_SIZE);
    dim3 gridDim(num_blocks_x, num_blocks_y);

    matrix_transpose_tiled_kernel<T, TILE_SIZE><<< gridDim, blockDim >>>(out, in);
    hipDeviceSynchronize();
  #endif
}

template <typename T>
void matrix_mult_seq(matrix_t<T> a, matrix_t<T> b, matrix_t<T> r) {
  for (int i = 0; i < r.height; ++i) {
    for (int j = 0; j < r.width; ++j) {
      T res = 0;
      for (int k = 0; k < a.width; ++k) {
        res += a.elements[i * a.width + k] * b.elements[k * b.width + j];
      }
      r.elements[i * r.width + j] = res;
    }
  }
}

template <typename T>
void matrix_mult_cuda_naive(const unsigned int block_size, matrix_t<T> a, matrix_t<T> b, matrix_t<T> r) {
  #ifdef __HIPCC__
  // Allocate and specify dimensions
  unsigned int num_blocks_x, num_blocks_y;

  num_blocks_x = ((r.width % block_size) == 0) ?
                   r.width / block_size     :
                   r.width / block_size + 1 ;

  num_blocks_y = ((r.height % block_size) == 0) ?
                   r.height / block_size     :
                   r.height / block_size + 1 ;

  dim3 blockDim(block_size, block_size);
  dim3 gridDim(num_blocks_x, num_blocks_y);
  // Invoke the kernel
  matrix_mult_naive_kernel<<<gridDim, blockDim>>>(a,b,r);
  hipDeviceSynchronize(); 
  #endif
}

template <typename T>
void matrix_mult_cuda_tiled(matrix_t<T> a, matrix_t<T> b, matrix_t<T> r) {
  #ifdef __HIPCC__
  unsigned int block_size = TILE_SIZE;
  // Allocate and specify dimensions
  unsigned int num_blocks_x, num_blocks_y;

  num_blocks_x = ((r.width % block_size) == 0) ?
                   r.width / block_size     :
                   r.width / block_size + 1 ;

  num_blocks_y = ((r.height % block_size) == 0) ?
                   r.height / block_size     :
                   r.height / block_size + 1 ;

  dim3 blockDim(block_size, block_size);
  dim3 gridDim(num_blocks_x, num_blocks_y);
  // Invoke the kernel
  matrix_mult_tiled_kernel<T, TILE_SIZE><<<gridDim, blockDim>>>(a,b,r);
  hipDeviceSynchronize(); 
  #endif
}

int timeval_subtract(struct timeval *result, struct timeval *t2, struct timeval *t1)
{
    unsigned int resolution=1000000;
    long int diff = (t2->tv_usec + resolution * t2->tv_sec) - (t1->tv_usec + resolution * t1->tv_sec);
    result->tv_sec = diff / resolution;
    result->tv_usec = diff % resolution;
    return (diff<0);
}

void timer_start() {
  gettimeofday(&t_start, NULL);
}

unsigned long int timer_stop() {
  unsigned long int elapsed;
  gettimeofday(&t_end, NULL);
  timeval_subtract(&t_diff, &t_end, &t_start);
  elapsed = (t_diff.tv_sec*1e6+t_diff.tv_usec);
  return elapsed;
}

int main(int argc, char *argv[]) {
  unsigned long int elapsed;
  // Create input matrix
  matrix_t<float> m_in;
  m_in.width = MATRIX_SIZE;
  m_in.height = MATRIX_SIZE;
  m_in.elements = (float*) malloc(m_in.width * m_in.height * sizeof(float));
  matrix_fill_random_float(m_in,0.0,10.0);

  // Transpose using sequential implementation
  matrix_t<float> m_out_seq;
  m_out_seq.width = m_in.height;
  m_out_seq.height = m_in.width;
  m_out_seq.elements = (float*) malloc(m_out_seq.width * m_out_seq.height * sizeof(float));
  timer_start();
  matrix_transpose_seq<float>(m_out_seq, m_in);
  elapsed = timer_stop();
  printf("Sequential implementation of transpose finished in %lu microseconds!\n", elapsed);

  // Transpose using OMP implementation
  #if defined(_OPENMP)
  matrix_t<float> m_out_omp;
  m_out_omp.width = m_in.height;
  m_out_omp.height = m_in.width;
  m_out_omp.elements = (float*) malloc(m_out_omp.width * m_out_omp.height * sizeof(float));
  timer_start();
  matrix_transpose_omp<float>(m_out_omp, m_in);
  elapsed = timer_stop();
  if (matrix_is_equal(m_out_seq, m_out_omp)) {
    printf("OMP implementation of transpose produced the CORRECT result in %lu microseconds!\n", elapsed);
  } else {
    printf("OMP implementation of transpose produced an INCORRECT result in %lu microseconds!\n", elapsed);
  }
  free(m_out_omp.elements);
  #else
  printf("OMP not supported by the current compiler... Skipping...\n");
  #endif

  // Transpose using naive CUDA implementation
  #ifdef __HIPCC__
  // Device structs
  matrix_t<float> d_m_out_cuda_naive, d_m_in_cuda_naive, m_out_cuda_naive;
  d_m_in_cuda_naive.width   = m_in.width;
  d_m_in_cuda_naive.height  = m_in.height;
  d_m_out_cuda_naive.width  = m_in.height;
  d_m_out_cuda_naive.height = m_in.width;
  m_out_cuda_naive.width    = m_in.height;
  m_out_cuda_naive.height   = m_in.width;
  m_out_cuda_naive.elements = (float*) malloc(
    m_out_cuda_naive.width * m_out_cuda_naive.height * sizeof(float)
  );
  // Copy input array to device
  hipMalloc(
    (void**) &(d_m_in_cuda_naive.elements),
    d_m_in_cuda_naive.width * d_m_in_cuda_naive.height * sizeof(float)
  );
  hipMalloc(
    (void**) &(d_m_out_cuda_naive.elements),
    d_m_out_cuda_naive.width * d_m_out_cuda_naive.height * sizeof(float)
  );
  hipMemcpy(
    d_m_in_cuda_naive.elements, m_in.elements,
    m_in.width * m_in.height * sizeof(float),
    hipMemcpyHostToDevice
  );
  timer_start();
  matrix_transpose_cuda_naive<float>(512, d_m_out_cuda_naive, d_m_in_cuda_naive);
  elapsed = timer_stop();
  hipMemcpy(
    m_out_cuda_naive.elements, d_m_out_cuda_naive.elements,
    d_m_out_cuda_naive.width * d_m_out_cuda_naive.height * sizeof(float),
    hipMemcpyDeviceToHost
  );
  printf("\nInput array:\n");
  print_matrix(m_in);
  printf("\nSequential output array:\n");
  print_matrix(m_out_seq);
  printf("\nCuda output array:\n");
  print_matrix(m_out_cuda_naive);
  if (matrix_is_equal(m_out_seq, m_out_cuda_naive)) {
    printf("CUDA implementation (naive) of transpose produced the CORRECT result in %lu microseconds!\n", elapsed);
  } else {
    printf("CUDA implementation (naive) of transpose produced an INCORRECT result in %lu microseconds!\n", elapsed);
  }
  hipFree(d_m_in_cuda_naive.elements);
  hipFree(d_m_out_cuda_naive.elements);
  free(m_out_cuda_naive.elements);
  #else
  printf("CUDA not supported by the current compiler... Skipping...\n");
  #endif

  // Transpose using tiled CUDA implementation
  #ifdef __HIPCC__
  // Device structs
  matrix_t<float> d_m_out_cuda_tiled, d_m_in_cuda_tiled, m_out_cuda_tiled;
  d_m_in_cuda_tiled.width   = m_in.width;
  d_m_in_cuda_tiled.height  = m_in.height;
  d_m_out_cuda_tiled.width  = m_in.height;
  d_m_out_cuda_tiled.height = m_in.width;
  m_out_cuda_tiled.width    = m_in.height;
  m_out_cuda_tiled.height   = m_in.width;
  m_out_cuda_tiled.elements = (float*) malloc(
    m_out_cuda_tiled.width * m_out_cuda_tiled.height * sizeof(float)
  );
  // Copy input array to device
  hipMalloc(
    (void**) &(d_m_in_cuda_tiled.elements),
    d_m_in_cuda_tiled.width * d_m_in_cuda_tiled.height * sizeof(float)
  );
  hipMalloc(
    (void**) &(d_m_out_cuda_tiled.elements),
    d_m_out_cuda_tiled.width * d_m_out_cuda_tiled.height * sizeof(float)
  );
  hipMemcpy(
    d_m_in_cuda_tiled.elements, m_in.elements,
    d_m_in_cuda_tiled.width * d_m_in_cuda_tiled.height * sizeof(float),
    hipMemcpyHostToDevice
  );
  timer_start();
  matrix_transpose_cuda_tiled<float>(d_m_out_cuda_tiled, d_m_in_cuda_tiled);
  elapsed = timer_stop();
  hipMemcpy(
    m_out_cuda_tiled.elements, d_m_out_cuda_tiled.elements,
    d_m_out_cuda_tiled.width * d_m_out_cuda_tiled.height * sizeof(float),
    hipMemcpyDeviceToHost
  );
  if (matrix_is_equal(m_out_seq, m_out_cuda_tiled)) {
    printf("CUDA implementation (tiled) of transpose produced the CORRECT result in %lu microseconds!\n", elapsed);
  } else {
    printf("CUDA implementation (tiled) of transpose produced an INCORRECT result in %lu microseconds!\n", elapsed);
  }
  hipFree(d_m_in_cuda_tiled.elements);
  hipFree(d_m_out_cuda_tiled.elements);
  free(m_out_cuda_tiled.elements);
  #else
  printf("CUDA not supported by the current compiler... Skipping...\n");
  #endif

  // Create input matrix
  matrix_t<float> m_in_2;
  m_in_2.width = MATRIX_SIZE;
  m_in_2.height = MATRIX_SIZE;
  m_in_2.elements = (float*) malloc(m_in_2.width * m_in_2.height * sizeof(float));
  matrix_fill_random_float(m_in_2,0.0,10.0);

  // Multiply using sequential implementation
  matrix_t<float> m_mult_out_seq;
  m_mult_out_seq.width  = m_in_2.width;
  m_mult_out_seq.height = m_in.height;
  m_mult_out_seq.elements = (float*) malloc(m_mult_out_seq.width * m_mult_out_seq.height * sizeof(float));
  timer_start();
  matrix_mult_seq<float>(m_in, m_in_2, m_mult_out_seq);
  elapsed = timer_stop();
  printf("Sequential implementation of multiply finished in %lu microseconds!\n", elapsed);
  
  // Multiply using naive CUDA implementation
  #ifdef __HIPCC__
  // Device structs
  matrix_t<float> m_mult_out_cuda_naive, d_m_mult_out_cuda_naive, d_m_mult_in_cuda_naive, d_m_mult_in_2_cuda_naive;
  d_m_mult_in_cuda_naive.width    = m_in.width;
  d_m_mult_in_cuda_naive.height   = m_in.height;
  d_m_mult_in_2_cuda_naive.width  = m_in_2.width;
  d_m_mult_in_2_cuda_naive.height = m_in_2.height;
  d_m_mult_out_cuda_naive.width   = m_in_2.width;
  d_m_mult_out_cuda_naive.height  = m_in.height;
  m_mult_out_cuda_naive.width    = m_in_2.width;
  m_mult_out_cuda_naive.height   = m_in.height;
  m_mult_out_cuda_naive.elements = (float*) malloc(
    m_mult_out_cuda_naive.width * m_mult_out_cuda_naive.height * sizeof(float)
  );
  // Copy input array to device
  hipMalloc(
    (void**) &(d_m_mult_in_cuda_naive.elements),
    d_m_mult_in_cuda_naive.width * d_m_mult_in_cuda_naive.height * sizeof(float)
  );
  hipMalloc(
    (void**) &(d_m_mult_in_2_cuda_naive.elements),
    d_m_mult_in_2_cuda_naive.width * d_m_mult_in_2_cuda_naive.height * sizeof(float)
  );
  hipMalloc(
    (void**) &(d_m_out_cuda_naive.elements),
    d_m_out_cuda_naive.width * d_m_out_cuda_naive.height * sizeof(float)
  );
  hipMemcpy(
    d_m_in_cuda_naive.elements, m_in.elements,
    d_m_in_cuda_naive.width * d_m_in_cuda_naive.height * sizeof(float),
    hipMemcpyHostToDevice
  );
  hipMemcpy(
    d_m_mult_in_2_cuda_naive.elements, m_in_2.elements,
    d_m_mult_in_2_cuda_naive.width * d_m_mult_in_2_cuda_naive.height * sizeof(float),
    hipMemcpyHostToDevice
  );
  timer_start();
  matrix_mult_cuda_naive<float>(512, d_m_mult_in_cuda_naive, d_m_mult_in_2_cuda_naive, d_m_mult_out_cuda_naive);
  elapsed = timer_stop();
  hipMemcpy(
    m_mult_out_cuda_naive.elements, d_m_mult_out_cuda_naive.elements,
    d_m_mult_out_cuda_naive.width * d_m_mult_out_cuda_naive.height * sizeof(float),
    hipMemcpyDeviceToHost
  );
  if (matrix_is_equal(m_mult_out_seq, m_mult_out_cuda_naive)) {
    printf("CUDA implementation (naive) of multiply produced the CORRECT result in %lu microseconds!\n", elapsed);
  } else {
    printf("CUDA implementation (naive) of multiply produced an INCORRECT result in %lu microseconds!\n", elapsed);
  }
  #else
  printf("CUDA not supported by the current compiler... Skipping...\n");
  #endif

  // Multiply using tiled CUDA implementation
  #ifdef __HIPCC__
  // Device structs
  matrix_t<float> m_mult_out_cuda_tiled, d_m_mult_out_cuda_tiled, d_m_mult_in_cuda_tiled, d_m_mult_in_2_cuda_tiled;
  d_m_mult_in_cuda_tiled.width    = m_in.width;
  d_m_mult_in_cuda_tiled.height   = m_in.height;
  d_m_mult_in_2_cuda_tiled.width  = m_in_2.width;
  d_m_mult_in_2_cuda_tiled.height = m_in_2.height;
  d_m_mult_out_cuda_tiled.width   = m_in_2.width;
  d_m_mult_out_cuda_tiled.height  = m_in.height;
  m_mult_out_cuda_tiled.width    = m_in_2.width;
  m_mult_out_cuda_tiled.height   = m_in.height;
  m_mult_out_cuda_tiled.elements = (float*) malloc(
    m_mult_out_cuda_tiled.width * m_mult_out_cuda_tiled.height * sizeof(float)
  );
  // Copy input array to device
  hipMalloc(
    (void**) &(d_m_mult_in_cuda_tiled.elements),
    d_m_mult_in_cuda_tiled.width * d_m_mult_in_cuda_tiled.height * sizeof(float)
  );
  hipMalloc(
    (void**) &(d_m_mult_in_2_cuda_tiled.elements),
    d_m_mult_in_2_cuda_tiled.width * d_m_mult_in_2_cuda_tiled.height * sizeof(float)
  );
  hipMalloc(
    (void**) &(d_m_out_cuda_tiled.elements),
    d_m_out_cuda_tiled.width * d_m_out_cuda_tiled.height * sizeof(float)
  );
  hipMemcpy(
    d_m_in_cuda_tiled.elements, m_in.elements,
    d_m_in_cuda_tiled.width * d_m_in_cuda_tiled.height * sizeof(float),
    hipMemcpyHostToDevice
  );
  hipMemcpy(
    d_m_mult_in_2_cuda_tiled.elements, m_in_2.elements,
    d_m_mult_in_2_cuda_tiled.width * d_m_mult_in_2_cuda_tiled.height * sizeof(float),
    hipMemcpyHostToDevice
  );
  timer_start();
  matrix_mult_cuda_tiled<float>(d_m_mult_in_cuda_tiled, d_m_mult_in_2_cuda_tiled, d_m_mult_out_cuda_tiled);
  elapsed = timer_stop();
  hipMemcpy(
    m_mult_out_cuda_tiled.elements, d_m_mult_out_cuda_tiled.elements,
    d_m_mult_out_cuda_tiled.width * d_m_mult_out_cuda_tiled.height * sizeof(float),
    hipMemcpyDeviceToHost
  );
  if (matrix_is_equal(m_mult_out_seq, m_mult_out_cuda_tiled)) {
    printf("CUDA implementation (tiled) of multiply produced the CORRECT result in %lu microseconds!\n", elapsed);
  } else {
    printf("CUDA implementation (tiled) of multiply produced an INCORRECT result in %lu microseconds!\n", elapsed);
  }
  #else
  printf("CUDA not supported by the current compiler... Skipping...\n");
  #endif
  return 0;
}
